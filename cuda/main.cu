
#include <hip/hip_runtime.h>
#include <cstdlib>
#include <iostream>
#include <string.h>
#include <stdint.h>
#include <sys/time.h>

// Utils

__device__ uint random_uint(uint seed) {
    uint x = seed;
    x ^= x << 13;
    x ^= x >> 17;
    x ^= x << 5;
    return x;
}

double cpuSecond() {
    struct timeval tp;
    gettimeofday(&tp,NULL);
    return ((double)tp.tv_sec + (double)tp.tv_usec*1.e-6);
}

typedef struct {
    int x;
    int y;
    int state;
    uint seed;
} Particle;


__device__ int d_width;
__device__ int d_height;
__device__ int* d_map;

__global__ void init_particles(Particle* particles, int count) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < count) {
        Particle p;
        p.seed = (uint) (idx * 4);
        p.state = 0;
        p.x = (int)(p.seed % d_width);
        p.seed = random_uint(p.seed);
        p.y = (int)(p.seed % d_height);
        particles[idx] = p;
        // printf("%d) %d (%d, %d)\n", idx, p.seed, p.x, p.y);
    }
}

__device__ void crystallize(Particle* p) {
    p->state = 1;
    int i = p->y * d_width + p->x;
    atomicAdd(&d_map[i], 1);
}

__device__ bool should_crystallize(int x, int y) {
    for (int xs = -1; xs <= 1; xs++) {
        for (int ys = -1; ys <= 1; ys++) {
            if (ys == 0 && xs == 0) {
                continue;
            }
            if (xs + x < 0 || xs + x >= d_width) {
                continue;
            }
            if (ys + y < 0 || ys + y >= d_height) {
                continue;
            }
            int j = ((y + ys) * d_width) + (x + xs);
            if (d_map[j] > 0) { // Found a Crystal
                return true;
            }
        }
    }
    return false;
}

__device__ Particle move_particle(Particle particle) {
    particle.seed = random_uint(particle.seed);
    int mov = particle.seed % 4;
    switch (mov) {
        case 0: 
            particle.x += 1;
            break;
        case 1: 
            particle.x -= 1;
            break;
        case 2: 
            particle.y += 1;
            break;
        case 3: 
            particle.y -= 1;
            break;
    }
    // Check Bounds
    if (particle.x >= d_width) {
        particle.x = 0;
    } else if (particle.x < 0) {
        particle.x = d_width - 1;
    }
    if (particle.y >= d_height) {
        particle.y = 0;
    } else if (particle.y < 0) {
        particle.y = d_height - 1;
    }
    return particle;
}

__global__ void tick(Particle* particles, int count) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < count) {
        Particle p = particles[idx];
        if (p.state == 1) { // Crystallized Particle
            return;
        }
        if (should_crystallize(p.x, p.y)) {
            particles[idx].state = 1;
            int i = p.y * d_width + p.x;
            atomicAdd(&d_map[i], 1);
        } else {
            p = move_particle(p);
            particles[idx] = p;
        }
    }
}

void cuda_error() {
    hipError_t err = hipGetLastError();
    if (err != hipSuccess) {
        while (err != hipSuccess) {
            printf("Error:\n\t%s\n", hipGetErrorString(err));
            err = hipGetLastError();
        }
        exit(EXIT_FAILURE);
    }
}

void save_map(char* file_name, int* map, int width, int height) {
    FILE *fp = fopen(file_name, "w");
    if (fp == NULL) {
        printf("Error opening the file %s", file_name);
        return;
    }
    for (int i = 0; i < width * height; i++) {
        if (i % width == 0) {
            fprintf(fp, "\n");
        }
        fprintf(fp, "%d ", map[i]);
    }
    fclose(fp);
}

int main(int argc, char* argv[]) {
    // Read Arguments
    if (argc < 6) {
        printf("Usage: main WIDTH HEIGHT PARTICLES STEPS X Y out_map\n");
        exit(EXIT_FAILURE);
    }
    int h_width = atoi(argv[1]);
    int h_height = atoi(argv[2]);
    int h_particles_count = atoi(argv[3]);
    int h_steps = atoi(argv[4]);
    int h_start_x = atoi(argv[5]);
    int h_start_y = atoi(argv[6]);
    int h_start_idx = h_start_y * h_width + h_start_x;
    char* h_out_map = "crystal.txt";
    if (argc >= 8) {
        h_out_map = argv[7];
    }
    double iStart = cpuSecond();
    // Init Map
    int* h_map = (int*)malloc(sizeof(int) * h_width * h_height);
    for (int i = 0; i < h_width * h_height; i++) {
        h_map[i] = 0;
    }
    h_map[h_start_idx] = 1; // First Crystal
    // Allocating Device Memory
    // Initializing Devices
    int* d_map_tmp;
    hipMalloc((void**)&d_map_tmp, sizeof(int) * h_width * h_height);
    hipMemcpy(d_map_tmp, h_map, sizeof(int) * h_width * h_height, hipMemcpyHostToDevice);
    hipMemcpyToSymbol(HIP_SYMBOL(d_width), &h_width, sizeof(int));
    hipMemcpyToSymbol(HIP_SYMBOL(d_height), &h_height, sizeof(int));
    hipMemcpyToSymbol(HIP_SYMBOL(d_map), &d_map_tmp, sizeof(int*));
    Particle* d_particles;
    hipMalloc((void**)&d_particles, sizeof(int) * h_particles_count);
    cuda_error();
    // Initializing Particles
    const int base_div = 256; 
    int init_threads = base_div + (h_particles_count % base_div);
    int init_blocks = h_particles_count / base_div;
    if (h_particles_count < 256) {
        init_threads = h_particles_count;
        init_blocks = 1;
    }
    init_particles<<<init_blocks, init_threads>>>(d_particles, h_particles_count);
    hipDeviceSynchronize();
    cuda_error();
	printf("Initialization time: %.3f milliseconds\n", 1000*(cpuSecond() - iStart));
    printf("Running Simulation\n");
    double sStart = cpuSecond();
    for (int step = 0; step < h_steps; step++) {
        // printf("Step %d\n", step);
        tick<<<init_blocks, init_threads>>>(d_particles, h_particles_count);
        // cudaDeviceSynchronize();
        cuda_error();
    }
    hipDeviceSynchronize();
    cuda_error();
	printf("Simulation time: %.3f milliseconds\n", 1000*(cpuSecond() - sStart));
    hipMemcpyFromSymbol(&d_map_tmp, HIP_SYMBOL(d_map), sizeof(int *));
    hipMemcpy(h_map, d_map_tmp, sizeof(int) * h_width * h_height, hipMemcpyDeviceToHost);
    cuda_error();
    // Save Map 
    save_map(h_out_map, h_map, h_width, h_height);
    // Free Memory
    hipFree(d_map_tmp);
    hipFree(d_particles);
    free(h_map);
    cuda_error();
	printf("Total time: %.3f milliseconds\n", 1000*(cpuSecond() - iStart));
    return 0;
}